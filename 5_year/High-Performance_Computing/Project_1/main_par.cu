/*
 * Based on materials from:
 * https://github.com/csc-training/openacc/tree/master/exercises/heat
 * https://enccs.github.io/OpenACC-CUDA-beginners/2.02_cuda-heat-equation/
 * changed 23 nov 2022 - vad@fct.unl.pt
 */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>

#ifdef PNG
#include "pngwriter.h"
#endif

#define THREADS_BLOCK 8

/* Convert 2D index layout to unrolled 1D layout
 * \param[in] i      Row index
 * \param[in] j      Column index
 * \param[in] width  The width of the area
 * \returns An index in the unrolled 1D array.
 */
int __host__ __device__ getIndex(const int i, const int j, const int width)
{
    return i*width + j;
}

double timedif(struct timespec *t, struct timespec *t0) {
    return (t->tv_sec-t0->tv_sec)+1.0e-9*(double)(t->tv_nsec-t0->tv_nsec);
}

void initTemp(float *T, int h, int w) {
    // Initializing the data with heat from top side
    // all other points at zero
    for (int i = 0; i < w; i++)
    {
        T[i] = 100.0;
    }
}

/* write_pgm - write a PGM image ascii file
 */
void write_pgm(FILE *f, float *img, int width, int height, int maxcolors) {
    // header
    fprintf(f, "P2\n%d %d %d\n", width, height, maxcolors);
    // data
    for (int l = 0; l < height; l++) {
        for (int c = 0; c < width; c++) {
            int p = (l * width + c);
            fprintf(f, "%d ", (int)(img[p]));
        }
        putc('\n', f);
    }
}


/* write heat map image
*/
void writeTemp(float *T, int h, int w, int n) {
    char filename[64];
#ifdef PNG
    sprintf(filename, "P_heat_%06d.png", n);
    save_png(T, h, w, filename, 'c');
#else
    sprintf(filename, "P_heat_%06d.pgm", n);
    FILE *f=fopen(filename, "w");
    write_pgm(f, T, w, h, 100);
    fclose(f);
#endif
}

__global__ void computeHeatShared(float *T, float *Tp, int nx, int  ny, float a, float dt, float h2)
{
    __shared__ float s_T[(THREADS_BLOCK + 2)*(THREADS_BLOCK + 2)];
    int row = threadIdx.x + blockIdx.x*blockDim.x;
    int col = threadIdx.y + blockIdx.y*blockDim.y;

    int s_row = threadIdx.x + 1;
    int s_col = threadIdx.y + 1;
    int s_ny = THREADS_BLOCK + 2;

    // Load data into shared memory
    // Central square
    s_T[getIndex(s_row, s_col, s_ny)] = T[getIndex(row, col, ny)];
    // Top border
    if (s_row == 1 && row != 0)
    {
        s_T[getIndex(0, s_col, s_ny)] = T[getIndex(blockIdx.x*blockDim.x - 1, col, ny)];
    }
    // Bottom border
    if (s_row == THREADS_BLOCK && row != nx - 1)
    {
        s_T[getIndex(THREADS_BLOCK + 1, s_col, s_ny)] = T[getIndex((blockIdx.x + 1)*blockDim.x, col, ny)];
    }
    // Left border
    if ( s_col == 1 && col != 0)
    {
        s_T[getIndex(s_row, 0, s_ny)] = T[getIndex(row, blockIdx.y*blockDim.y - 1, ny)];
    }
    // Right border
    if (s_col == THREADS_BLOCK && col != ny - 1)
    {
        s_T[getIndex(s_row, THREADS_BLOCK + 1, s_ny)] = T[getIndex(row, (blockIdx.y + 1)*blockDim.y, ny)];
    }

    // Make sure all the data is loaded before computing
    __syncthreads();

    if(col < nx-1 && row < ny-1 && col > 0 && row > 0)
        {
            float tij = s_T[getIndex(s_row, s_col, s_ny)];
            float tim1j = s_T[getIndex(s_row-1, s_col, s_ny)];
            float tijm1 = s_T[getIndex(s_row, s_col-1, s_ny)];
            float tip1j = s_T[getIndex(s_row+1, s_col, s_ny)];
            float tijp1 = s_T[getIndex(s_row, s_col+1, s_ny)];

            // Explicit scheme
            Tp[getIndex(row,col,ny)] = tij + a * dt * ( (tim1j + tip1j + tijm1 + tijp1 - 4.0*tij)/h2 );

    }
}

__global__ void computeHeat(float *T, float *Tp, int nx, int  ny, float a, float dt, float h2) {
    int row = threadIdx.x + blockIdx.x*blockDim.x;
    int col = threadIdx.y + blockIdx.y*blockDim.y;

    int index = row*ny + col;
    if(col < nx-1 && row < ny-1 && col > 0 && row > 0){
       	float tij = T[index];
	float tim1j = T[(row-1)*ny + col];
	float tijm1 = T[row*ny + (col-1)];
	float tip1j = T[(row + 1)*ny + col];
	float tijp1 = T[row*ny + (col+1)];
	Tp[index] = tij + a * dt * ( (tim1j + tip1j + tijm1 + tijp1 - 4.0*tij)/h2 );
    }


}



int main(int argc, char *argv[])
{

    int shared = atoi(argv[1]);
    const int nx = 200; // 200;   // Width of the area
    const int ny = 200; // 200;   // Height of the area

    const float a = 0.5;     // Diffusion constant

    const float h = 0.005; // 0.005;   // h=dx=dy  grid spacing

    const float h2 = h*h;

    const float dt =  h2 / (4.0 * a); // Largest stable time step
    const int numSteps = 100000;      // Number of time steps to simulate (time=numSteps*dt)
    const int outputEvery = 10000;   // How frequently to write output image

    int numElements = nx*ny;

    // Allocate two sets of data for current and next timesteps
    float* Tn   = (float*)malloc(numElements * sizeof(float));

    // Initializing the data for T0
    initTemp(Tn, nx, ny);

    // Fill in the data on the next step to ensure that the boundaries are identical.

    printf("Simulated time: %g (%d steps of %g)\n", numSteps*dt, numSteps, dt);
    printf("Simulated surface: %gx%g (in %dx%g divisions)\n", nx*h, ny*h, nx, h);
    writeTemp(Tn, nx, ny, 0);

    dim3 dimBlock(THREADS_BLOCK,THREADS_BLOCK,1); // 32*32THREADS_BLOCK);
    dim3 dimGrid(((nx+dimBlock.x-1)/dimBlock.x), ((ny+dimBlock.y-1)/dimBlock.y),1);

    float *cuda_tn;
    hipMalloc(&cuda_tn, numElements*sizeof(float));
    float *cuda_tnp1;
    hipMalloc(&cuda_tnp1, numElements*sizeof(float));

    if ( cuda_tn==NULL || cuda_tnp1==NULL ) {
        fprintf(stderr,"No GPU mem!\n");
        return EXIT_FAILURE;
    }

    //timing
    struct timespec t0, t;
    clock_gettime(CLOCK_MONOTONIC, &t0);

  hipMemcpy(cuda_tnp1, Tn, numElements*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cuda_tn, Tn, numElements*sizeof(float),hipMemcpyHostToDevice);

    // Main loop
    for (int n = 0; n <= numSteps; n++)
    {
        if(shared){
            computeHeatShared<<<dimGrid, dimBlock>>>(cuda_tn, cuda_tnp1, nx,ny,a,dt,h2);
        }else{
            computeHeat<<<dimGrid, dimBlock>>>(cuda_tn, cuda_tnp1, nx,ny,a,dt,h2);
        }
        
        // Write the output if needed
        if ((n+1) % outputEvery == 0) {
        hipMemcpy(Tn, cuda_tnp1, numElements * sizeof(float), hipMemcpyDeviceToHost);
            hipError_t err=hipGetLastError();
            if (err!=hipSuccess) {
                fprintf(stderr, "err=%u %s\n%s\n", (unsigned) err, hipGetErrorString(err),
                        "Problems executing kernel");
                exit(1);
            }
            writeTemp(Tn, nx, ny, n + 1);
        }

        // Swapping the pointers for the next timestep
        float* t = cuda_tn;
        cuda_tn = cuda_tnp1;
        cuda_tnp1 = t;
    }

    // Timing
    clock_gettime(CLOCK_MONOTONIC, &t);
    printf("time: %f seconds\n", timedif(&t, &t0) );

    // Release the memory
    free(Tn);
    hipFree(cuda_tn);
    hipFree(cuda_tnp1);

    return 0;
}
